#include "hip/hip_runtime.h"
// Copyright (2019-2022) Paul Scherrer Institute
// SPDX-License-Identifier: GPL-3.0-or-later

#include "PedestalCalcGPU.h"
#include "JFJochException.h"
#include "PedestalCalcCPU.h"

struct CudaStreamWrapper {
    hipStream_t v;
};

inline void cuda_err(hipError_t val) {
    if (val != hipSuccess)
        throw JFJochException(JFJochExceptionCategory::GPUCUDAError, hipGetErrorString(val));
}

PedestalCalcGPU::PedestalCalcGPU(const DiffractionExperiment & experiment, int64_t in_lines, int32_t gpu_device) :
        PedestalCalc(experiment, in_lines) {

    int device_count;
    cuda_err(hipGetDeviceCount(&device_count));

    if (device_count == 0)
        throw JFJochException(JFJochExceptionCategory::GPUCUDAError, "No CUDA devices found");

    if (gpu_device < 0)
        gpu_device = threadid++;

    if (device_count > 1)
        cuda_err(hipSetDevice(gpu_device % device_count));

    int deviceId;
    cuda_err(hipGetDevice(&deviceId));
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    cudastream = new(CudaStreamWrapper);
    cuda_err(hipStreamCreate(&cudastream->v));

    cuda_err(hipMallocManaged(&currPedestal, lines * RAW_MODULE_COLS * sizeof(double)));
    cuda_err(hipMallocManaged(&currPedestalSquare, lines * RAW_MODULE_COLS * sizeof(double)));
    cuda_err(hipMallocManaged(&wrongCount, lines * RAW_MODULE_COLS * sizeof(uint32_t)));
    cuda_err(hipMalloc(&gpu_image, lines * RAW_MODULE_COLS * sizeof(uint16_t)));

    for (int i = 0; i < lines * RAW_MODULE_COLS; i++) {
        currPedestal[i] = 0;
        currPedestalSquare[i] = 0;
        wrongCount[i] = 0;
    }
}

PedestalCalcGPU::~PedestalCalcGPU() {

    hipStreamDestroy(cudastream->v);
    delete(cudastream);

    hipFree(currPedestal);
    hipFree(currPedestalSquare);
    hipFree(wrongCount);
    hipFree(gpu_image);
}

template <unsigned int GAIN_BIT> __global__
void analyze_pixel_add(const uint16_t *raw_image, double *currPedestal, double *currPedestalSquare, uint32_t *wrongCount,
                  uint32_t lines) {

    uint32_t pixel = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t line = pixel / RAW_MODULE_COLS;

    if (line < lines) {
        float adc = raw_image[pixel] & 0x3FFF;
        uint16_t gain = raw_image[pixel] & 0xc000;

        if (gain != GAIN_BIT)
            wrongCount[pixel]++;

        currPedestal[pixel] += adc;
        currPedestalSquare[pixel] += adc * adc;

    }
}

template <unsigned int GAIN_BIT, unsigned int WINDOW_SIZE> __global__
void analyze_pixel_avg(const uint16_t *raw_image, double *currPedestal, double *currPedestalSquare, uint32_t *wrongCount,
                       uint32_t lines) {

    uint32_t pixel = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t line = pixel / RAW_MODULE_COLS;

    if (line < lines) {
        float adc = (raw_image[pixel] & 0x3FFF);
        uint16_t gain = raw_image[pixel] & 0xc000;

        if (gain != GAIN_BIT)
            wrongCount[pixel]++;
        else {
            // Don't include wrong gains into moving average
            currPedestal[pixel] += adc - currPedestal[pixel] / WINDOW_SIZE;
            currPedestalSquare[pixel] += adc * adc - currPedestalSquare[pixel] / WINDOW_SIZE;
        }
    }
}

void PedestalCalcGPU::AnalyzeImage(const uint16_t *raw_image) {
    cuda_err(hipMemcpy(gpu_image, raw_image, lines * RAW_MODULE_COLS * sizeof(uint16_t), hipMemcpyHostToDevice));
    size_t nThreads = 256;
    size_t nBlocks  = ((lines * RAW_MODULE_COLS) / (nThreads * numberOfSMs) + 1) * numberOfSMs;

    switch (gain_level) {
        case 0:
            if (image_number < window_size)
                analyze_pixel_add<0><<<nBlocks, nThreads, 0, cudastream->v>>>
                    (gpu_image, currPedestal, currPedestalSquare, wrongCount, lines);
            else
                analyze_pixel_avg<0, PEDESTAL_WINDOW_SIZE><<<nBlocks, nThreads, 0, cudastream->v>>>
                        (gpu_image, currPedestal, currPedestalSquare, wrongCount, lines);
            break;
        case 1:
            if (image_number < window_size)
                analyze_pixel_add<0x4000><<<nBlocks, nThreads, 0, cudastream->v>>>
                        (gpu_image, currPedestal, currPedestalSquare, wrongCount, lines);
            else
                analyze_pixel_avg<0x4000, PEDESTAL_WINDOW_SIZE><<<nBlocks, nThreads, 0, cudastream->v>>>
                        (gpu_image, currPedestal, currPedestalSquare, wrongCount, lines);
            break;
        case 2:
            if (image_number < window_size)
                analyze_pixel_add<0xc000><<<nBlocks, nThreads, 0, cudastream->v>>>
                        (gpu_image, currPedestal, currPedestalSquare, wrongCount, lines);
            else
                analyze_pixel_avg<0xc000, PEDESTAL_WINDOW_SIZE><<<nBlocks, nThreads, 0, cudastream->v>>>
                        (gpu_image, currPedestal, currPedestalSquare, wrongCount, lines);
            break;
        default:
            break;
    }
    cuda_err(hipStreamSynchronize(cudastream->v));
    image_number++;
}

std::atomic<uint16_t> PedestalCalcGPU::threadid{0};

bool PedestalCalcGPU::GPUPresent() {
    int device_count;
    cuda_err(hipGetDeviceCount(&device_count));

    return (device_count > 0);
}

std::unique_ptr<PedestalCalc> MakePedestalCalc(const DiffractionExperiment& experiment, int64_t lines, int32_t thread_id) {
    if (PedestalCalcGPU::GPUPresent())
        return std::make_unique<PedestalCalcGPU>(experiment, lines, thread_id);
    else
        return std::make_unique<PedestalCalcCPU>(experiment, lines);
}