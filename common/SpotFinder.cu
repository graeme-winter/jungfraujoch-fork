#include "hip/hip_runtime.h"
// Copyright (2019-2022) Paul Scherrer Institute
// SPDX-License-Identifier: GPL-3.0-or-later

#include "SpotFinder.h"
#include "JFJochException.h"
#include <sstream>

// input      X x Y pixels array
// output     X x Y bit array

static constexpr int WARP_SIZE = 32;    // assume warp size of 32 cuda threads per warp

struct CudaStreamWrapper {
    hipStream_t v;
};

inline void cuda_err(hipError_t val) {
    if (val != hipSuccess)
        throw JFJochException(JFJochExceptionCategory::GPUCUDAError, hipGetErrorString(val));
}

// Calculate byte size of output
// output_type: type of output array elements
// ptr: dummy pointer to make template type inference possible
// xpixels: number of bits (=pixel results) in x direction
// ypixels: number of bits (=pixel results) in y direction
// returns the size of the output (xpixels X ypixels bits) in bytes for an array with elements of type output_type
template <typename output_type>
static std::size_t output_byte_size(const output_type* ptr, int32_t xpixels, int32_t ypixels) noexcept {
    constexpr auto bit_group_size = sizeof(output_type) * 8;    // number of bits per output array element
    return ((xpixels * ypixels + bit_group_size - 1) / bit_group_size) * sizeof(output_type);
}

// Get the bit ptr[line, col] of a bit array with lines of size xpixels bits
// output_type: type of bit array elements
// ptr: pointer to array with bits stored in element type output_type
// xpixels: size in bits (=pixel results) of one line (=row of array)
// line: bit line (=row) number
// col: bit column number
// returns the bit ptr[line, col]
template <typename output_type>
static bool get_bit(const output_type* ptr, int32_t xpixels, int32_t line, int32_t col) noexcept {
    constexpr auto bit_group_size = sizeof(output_type) * 8;    // number of bits per output array element
    const auto global_bit_idx = line * xpixels + col;
    const auto element_idx = global_bit_idx / bit_group_size;
    const auto local_bit_idx = global_bit_idx % bit_group_size;
    return (ptr[element_idx] & (1 << local_bit_idx)) != 0;
}

// Write pixel results to bit array
// params: spot finding parameters
// out: pixel result bit array
// pixel: flat pixel index = bit index into bit array
// val: pixel result
// **NOTE**: assumes sizeof(*out) * 8 == WARP_SIZE
__device__ __forceinline__ void write_result(const spot_parameters& params, uint32_t* out, int32_t pixel, uint8_t val)
{
    static_assert(sizeof(*out) * 8 == WARP_SIZE, "Violation of essential implementation assumtion: WARP_SIZE must match output array element type bit size!");
    static constexpr unsigned ALL_THREADS = unsigned{-1};
    const int32_t laneid = threadIdx.x & (WARP_SIZE - 1);
    unsigned result = __ballot_sync(ALL_THREADS, val);
    const int32_t idx = pixel / WARP_SIZE;      // global uint32_t index
    const int32_t bit = pixel % WARP_SIZE;      // local bit index

    if ((bit >= laneid) & (laneid == 0)) {      // write to upper part of uint32_t
        result <<= bit;
        if (result)
            atomicOr(&out[idx], result);
    } else if ((bit < laneid) & (bit == 0)) {   // write to lower part of uint32_t
        result >>= laneid;
        if (result)
            atomicOr(&out[idx], result);
    }
}

// Determine if pixel could be a spot
// params: spot finding parameters
// val: pixel value
// sum: window sum
// sum2: window sum of squares
// count: window valid pixels count
// return the pixel result: 0-no spot / 1-spot candidate
__device__ __forceinline__ uint8_t pixel_result(const spot_parameters& params, const int64_t val, int64_t sum, int64_t sum2, int64_t count)
{
    sum -= val;
    sum2 -= val * val;
    count -= 1;

    const int64_t var = count * sum2 - (sum * sum);   // This should be divided by ((2*NBX+1) * (2*NBY+1)-1)*((2*NBX+1) * (2*NBY+1))
    const int64_t in_minus_mean = val * count - sum;  // Should be divided by ((2*NBX+1) * (2*NBY+1));

    const int64_t tmp1 = in_minus_mean * in_minus_mean * (count-1);
    const float tmp2 = (var * count) * params.strong_pixel_threshold2;
    const bool strong_pixel = (val >= params.count_threshold) & (in_minus_mean > 0) & (tmp1 > tmp2);

    return strong_pixel ? 1 : 0;
}

// Find pixels that could be spots
// in: image input values
// out: pixel result bit array, 1 bit per pixel (0:no/1:candidate spot)
// params: spot finding parameters
//
// The algorithm uses multiple waves (blockDim.y) that run over sections of rows.
// Each wave will write output at the back row and read input at the front row.
// Each wave is split into column output sections (blockDim.x)
// A wave section (block) is responsible for a particular row/column section and
// maintains sum/sum2/count values per column for the output row.
// Every cuda thread is associated with a particular column. The thread maintains
// the sum/sum2/count values in shared memory for it's column. To do this, the input
// pixel values for the hight of the aggregation window are saved in shared memory.
extern __shared__ int32_t shared_mem[];
__global__ void analyze_pixel(const int16_t *in, uint32_t *out, const spot_parameters params)
{
    // assumption: 2 * params.nby + 1 <= params.rows and 2 * params.nbx + 1 <= params.cols
    const int32_t window = 2 * (int)params.nby + 1;                                         // vertical window

    const int32_t writeSize = blockDim.x - 2 * params.nbx;                                  // output columns per block
    const int32_t cmin = blockIdx.x * writeSize;                                            // lowest output column
    const int32_t cmax = min(cmin + writeSize, static_cast<int32_t>(params.cols));     // past highest output column
    const int32_t col = cmin + threadIdx.x - params.nbx;                                    // thread -> column mapping
    const bool data_col = (col >= 0) & (col < static_cast<int32_t>(params.cols));           // read global mem
    const bool result_col = (col >= cmin) & (col < cmax);                                   // write result
    const int32_t nWaves = gridDim.y;                                                       // number of waves
    const int32_t rowsPerWave = (params.lines + nWaves - 1) / nWaves;                       // rows per wave
    const int32_t rmin = blockIdx.y * rowsPerWave;                                          // lowest result row for this wave
    const int32_t rmax = min(rmin + rowsPerWave, static_cast<int32_t>(params.lines));  // past highest result row for this wave

    const int32_t left = max(static_cast<int32_t>(threadIdx.x) - static_cast<int32_t>(params.nbx), 0); // leftmost column touched by this thread
    const int32_t right = min(static_cast<int32_t>(threadIdx.x) + static_cast<int32_t>(params.nbx) + 1, static_cast<int32_t>(params.cols)); // past rightmost column touched by this thread

    int32_t back = rmin;                                                                    // back of wave for writing
    int32_t front = max(back - static_cast<int32_t>(params.nby), 0);                   // front of wave for reading (needs to overtake back initially)

    int32_t* shared_sum = shared_mem;                                                       // shared buffer [blockDim.x]
    int32_t* shared_sum2 = &shared_sum[blockDim.x];                                         // shared buffer [blockDim.x]
    int16_t* shared_count = reinterpret_cast<int16_t*>(&shared_sum2[blockDim.x]);           // shared buffer [blockDim.x]
    int16_t* shared_val = &shared_count[blockDim.x];                                        // shared cyclic buffer [window, blockDim.x]

    int32_t total_sum;                                                                      // totals
    int32_t total_sum2;
    int32_t total_count;

    // initialize sum, sum2, count, val buffers
    const int16_t ini = params.min_viable_number - 1;                                       // value that is NOT counted
    shared_sum[threadIdx.x] = 0;                                                            // shared values without effect on totals
    shared_sum2[threadIdx.x] = 0;
    shared_count[threadIdx.x] = 0;
    for (int i=0; i<window; i++)
        shared_val[i * blockDim.x + threadIdx.x] = ini;

    // wave front up to rmin + nby + 1
    do {
        if (data_col) {     // read at the front end of the wave
            const int16_t val = in[front * params.cols + col];
            shared_val[(front % window) * blockDim.x + threadIdx.x] = val;
            if (val >= params.min_viable_number) {
                shared_sum[threadIdx.x] += val;
                shared_sum2[threadIdx.x] += val * val;
                shared_count[threadIdx.x] += 1;
            }
        }
        front++;
    } while (front < rmin + static_cast<int32_t>(params.nby) + 1);
    // wave front up to rmax
    do {
        __syncthreads();    // make others see the shared values
        uint8_t val = 0;
        if (result_col) {   // write at the back end of the wave
            total_sum = total_sum2 = total_count = 0;
            for (auto j = left; j < right; j++) {
                total_sum += shared_sum[j];
                total_sum2 += shared_sum2[j];
                total_count += shared_count[j];
            }
            val = pixel_result(params, shared_val[(back % window) * blockDim.x + threadIdx.x], total_sum, total_sum2, total_count);
        }
        write_result(params, out, back * params.cols + col, val);
        back++;
        __syncthreads();    // keep shared values until others have seen them
        if (data_col) {     // read at the front end of the wave
            int16_t cnt = 0;
            int16_t old = shared_val[(front % window) * blockDim.x + threadIdx.x];
            if (old < params.min_viable_number) {
                old = 0;    // no effect value
                cnt = 1;    // bring count to normal
            }
            int16_t val = in[front * params.cols + col];
            shared_val[(front % window) * blockDim.x + threadIdx.x] = val;
            if (val < params.min_viable_number) {
                val = 0;    // no effect value
                cnt -= 1;   // count diff from normal
            }
            shared_sum[threadIdx.x] += val - old;
            shared_sum2[threadIdx.x] += val * val - old * old;
            shared_count[threadIdx.x] += cnt;
        }
        front++;
    } while (front < rmax);
    // wave back up to rmax
    do {
        __syncthreads();    // make others see the shared values
        uint8_t val = 0;
        if (result_col) {   // write at the back end of the wave
            total_sum = total_sum2 = total_count = 0;
            for (auto j = left; j < right; j++) {
                total_sum += shared_sum[j];
                total_sum2 += shared_sum2[j];
                total_count += shared_count[j];
            }
            val = pixel_result(params, shared_val[(back % window) * blockDim.x + threadIdx.x], total_sum, total_sum2, total_count);
        }
        write_result(params, out, back * params.cols + col, val);
        back++;
        __syncthreads();    // keep shared values until others have seen them
        if (data_col) {     // read at the front end of the wave if possible
            int16_t cnt = -1; // normal count diff
            int16_t old = shared_val[(front % window) * blockDim.x + threadIdx.x];
            if (old < params.min_viable_number) {
                old = 0;    // no effect value
                cnt += 1;   // bring count to normal
            }
            int16_t val = 0;
            if (front < params.lines) {
                val = in[front * params.cols + col];
                if (val < params.min_viable_number)
                    val = 0;    // no effect value
                else
                    cnt += 1;   // count diff from normal
            }
            shared_sum[threadIdx.x] += val - old;
            shared_sum2[threadIdx.x] += val * val - old * old;
            shared_count[threadIdx.x] += cnt;
        }
        front++;
    } while (back < rmax);
}

__global__ void apply_pixel_mask(int16_t *image, const uint8_t *mask, int npixel) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < npixel) {
        if (mask[idx] == 0)
            image[idx] = INT16_MIN;
    }
}

SpotFinder::SpotFinder(int32_t in_xpixels, int32_t in_ypixels, int32_t gpu_device) :
        xpixels(in_xpixels), ypixels(in_ypixels), gpu_out(nullptr) {

    int device_count;
    cuda_err(hipGetDeviceCount(&device_count));

    if (device_count == 0)
        throw JFJochException(JFJochExceptionCategory::GPUCUDAError, "No CUDA devices found");

    if (gpu_device < 0)
        gpu_device = threadid++;

    if (device_count > 1)
        cuda_err(hipSetDevice(gpu_device % device_count));

    int deviceId;
    cuda_err(hipGetDevice(&deviceId));
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    {
        int warp_size;
        cuda_err(hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, deviceId));
    }

    cudastream = new(CudaStreamWrapper);

    cuda_err(hipStreamCreate(&cudastream->v));
    cuda_err(hipMalloc(&gpu_mask, xpixels * ypixels * sizeof(int8_t)));
    cuda_err(hipMalloc(&gpu_in, xpixels * ypixels * sizeof(int16_t)));
    cuda_err(hipMalloc(&gpu_out, output_byte_size(gpu_out, xpixels, ypixels)));
    cuda_err(hipHostAlloc(&host_out, output_byte_size(host_out, xpixels, ypixels), hipHostMallocPortable));
    cuda_err(hipMemsetAsync(gpu_mask, 1, xpixels*ypixels, cudastream->v));
}

SpotFinder::SpotFinder(const DiffractionExperiment &experiment, int32_t gpu_device) :
        SpotFinder(experiment.GetXPixelsNum(), experiment.GetYPixelsNum(), gpu_device) {}

SpotFinder::~SpotFinder() {
    hipStreamDestroy(cudastream->v);
    delete(cudastream);

    hipHostFree(host_out);
    hipFree(gpu_in);
    hipFree(gpu_out);
}

void SpotFinder::SetInputBuffer(void *ptr) {
    host_in = (int16_t *) ptr;
}

bool SpotFinder::GPUPresent() {
    int device_count;
    cuda_err(hipGetDeviceCount(&device_count));

    return (device_count > 0);
}

void SpotFinder::RunSpotFinder(const JFJochProtoBuf::DataProcessingSettings &settings) {
    // data_in is CUDA registered memory

    // Run COLSPOT (GPU version)
    spot_parameters spot_params;
    spot_params.strong_pixel_threshold2 = settings.signal_to_noise_threshold() * settings.signal_to_noise_threshold();
    spot_params.nbx = settings.local_bkg_size();
    spot_params.nby = settings.local_bkg_size();
    spot_params.lines = ypixels;
    spot_params.cols = xpixels;
    spot_params.count_threshold = settings.photon_count_threshold();
    spot_params.min_viable_number = INT16_MIN + 5;

    if (2 * spot_params.nbx + 1 > windowSizeLimit)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "nbx exceeds window size limit");
    if (2 * spot_params.nby + 1 > windowSizeLimit)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "nby exceeds window size limit");
    if (windowSizeLimit > numberOfCudaThreads)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "window size limit exceeds number of cuda threads");
    if (windowSizeLimit > spot_params.cols)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "window size limit exceeds number of columns");
    if (windowSizeLimit > spot_params.lines)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "window size limit exceeds number of lines");
    if (host_in == nullptr)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "Host/GPU buffer not defined");

    cuda_err(hipMemcpy(gpu_in, host_in, xpixels * ypixels * sizeof(int16_t),
                        hipMemcpyHostToDevice));
    {
        // Apply pixel mask
        const auto nblocks = xpixels*ypixels / numberOfCudaThreads
                             + ((xpixels*ypixels % numberOfCudaThreads == 0) ? 0 : 1);
        apply_pixel_mask<<<nblocks, numberOfCudaThreads, 0, cudastream->v>>>
                (gpu_in, gpu_mask, xpixels*ypixels);
    }

    { // call cuda kernel
        const auto nWriters = numberOfCudaThreads - 2 * spot_params.nby;
        const auto nBlocks = (spot_params.cols + nWriters - 1) / nWriters;
        const auto window = 2 * spot_params.nby + 1;
        const auto sharedSize = (2 * sizeof(int32_t) +            // sum, sum2
                                 (1 + window) * sizeof(int16_t)   // count, val
                                ) * numberOfCudaThreads;
        const dim3 blocks(nBlocks, numberOfWaves);

        cuda_err(hipMemsetAsync(gpu_out, 0, output_byte_size(gpu_out, xpixels, ypixels), cudastream->v));
        analyze_pixel<<<blocks, numberOfCudaThreads, sharedSize, cudastream->v>>>
                (gpu_in, gpu_out, spot_params);
    }

    cuda_err(hipMemcpyAsync(host_out, gpu_out, output_byte_size(gpu_out, xpixels, ypixels),
                             hipMemcpyDeviceToHost,cudastream->v));
}

void SpotFinder::GetResults(StrongPixelSet &pixel_set, int64_t image_number) {
    if (host_in == nullptr)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "Host/GPU buffer not defined");

    cuda_err(hipStreamSynchronize(cudastream->v));

    for (int line = 0; line < ypixels; line++) {
        for (int col = 0; col < xpixels; col++) {
            if (get_bit(host_out, xpixels, line, col))
                pixel_set.AddStrongPixel(col, line, image_number, host_in[xpixels * line + col]);
        }
    }
}

void SpotFinder::GetResults(const DiffractionExperiment &experiment,
                            const JFJochProtoBuf::DataProcessingSettings &settings,
                            std::vector<DiffractionSpot> &vec,
                            int64_t image_number) {
    StrongPixelSet pixel_set;
    GetResults(pixel_set, image_number);
    pixel_set.FindSpots(experiment, settings, vec);
}

void SpotFinder::LoadMask(const std::vector<uint8_t> &mask) {
    if (mask.size() != xpixels * ypixels)
        throw JFJochException(JFJochExceptionCategory::InputParameterInvalid, "Mismatch in mask size");
    hipMemcpy(gpu_mask, mask.data(), xpixels*ypixels, hipMemcpyHostToDevice);
}

std::atomic<uint16_t> SpotFinder::threadid{0};
