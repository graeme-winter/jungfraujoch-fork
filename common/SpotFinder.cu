#include "hip/hip_runtime.h"
// Copyright (2019-2022) Paul Scherrer Institute
// SPDX-License-Identifier: GPL-3.0-or-later

#include "SpotFinder.h"
#include "JFJochException.h"
#include <sstream>

// input      X x Y pixels array
// output     X x Y byte array with 1 or 0

struct CudaStreamWrapper {
    hipStream_t v;
};

inline void cuda_err(hipError_t val) {
    if (val != hipSuccess)
        throw JFJochException(JFJochExceptionCategory::GPUCUDAError, hipGetErrorString(val));
}

// Determine if pixel could be a spot
// params: spot finding parameters
// val: pixel value
// sum: window sum
// sum2: window sum of squares
// count: window valid pixels count
__device__ __forceinline__ uint8_t pixel_result(const spot_parameters& params, const int64_t val, int64_t sum, int64_t sum2, int64_t count)
{
    sum -= val;
    sum2 -= val * val;
    count -= 1;

    const int64_t var = count * sum2 - (sum * sum);   // This should be divided by ((2*NBX+1) * (2*NBY+1)-1)*((2*NBX+1) * (2*NBY+1))
    const int64_t in_minus_mean = val * count - sum;  // Should be divided by ((2*NBX+1) * (2*NBY+1));

    const int64_t tmp1 = in_minus_mean * in_minus_mean * (count-1);
    const float tmp2 = (var * count) * params.strong_pixel_threshold2;
    const bool strong_pixel = (val >= params.count_threshold) & (in_minus_mean > 0) & (tmp1 > tmp2);

    return strong_pixel ? 1 : 0;
}

// Find pixels that could be spots
// in: image input values
// out: boolean output values
// params: spot finding parameters
//
// The algorithm uses multiple waves (blockDim.y) that run over sections of rows.
// Each wave will write output at the back row and read input at the front row.
// Each wave is split into column output sections (blockDim.x)
// A wave section (block) is responsible for a particular row/column section and
// maintains sum/sum2/count values per column for the output row.
// Every cuda thread is associated with a particular column. The thread maintains
// the sum/sum2/count values in shared memory for it's column. To do this, the input
// pixel values for the hight of the aggregation window are saved in shared memory.
extern __shared__ int32_t shared_mem[];
__global__ void analyze_pixel(const int16_t *in, uint8_t *out, const spot_parameters params)
{
    // assumption: 2 * params.nby + 1 <= params.rows and 2 * params.nbx + 1 <= params.cols
    const int32_t window = 2 * (int)params.nby + 1;                                         // vertical window

    const int32_t writeSize = blockDim.x - 2 * params.nbx;                                  // output columns per block
    const int32_t cmin = blockIdx.x * writeSize;                                            // lowest output column
    const int32_t cmax = min(cmin + writeSize, static_cast<int32_t>(params.cols));     // past highest output column
    const int32_t col = cmin + threadIdx.x - params.nbx;                                    // thread -> column mapping
    const bool data_col = (col >= 0) & (col < static_cast<int32_t>(params.cols));           // read global mem
    const bool result_col = (col >= cmin) & (col < cmax);                                   // write result
    const int32_t nWaves = gridDim.y;                                                       // number of waves
    const int32_t rowsPerWave = (params.lines + nWaves - 1) / nWaves;                       // rows per wave
    const int32_t rmin = blockIdx.y * rowsPerWave;                                          // lowest result row for this wave
    const int32_t rmax = min(rmin + rowsPerWave, static_cast<int32_t>(params.lines));  // past highest result row for this wave

    const int32_t left = max(static_cast<int32_t>(threadIdx.x) - static_cast<int32_t>(params.nbx), 0); // leftmost column touched by this thread
    const int32_t right = min(static_cast<int32_t>(threadIdx.x) + static_cast<int32_t>(params.nbx) + 1, static_cast<int32_t>(params.cols)); // past rightmost column touched by this thread

    int32_t back = rmin;                                                                    // back of wave for writing
    int32_t front = max(back - static_cast<int32_t>(params.nby), 0);                   // front of wave for reading (needs to overtake back initially)

    int32_t* shared_sum = shared_mem;                                                       // shared buffer [blockDim.x]
    int32_t* shared_sum2 = &shared_sum[blockDim.x];                                         // shared buffer [blockDim.x]
    int16_t* shared_count = reinterpret_cast<int16_t*>(&shared_sum2[blockDim.x]);           // shared buffer [blockDim.x]
    int16_t* shared_val = &shared_count[blockDim.x];                                        // shared cyclic buffer [window, blockDim.x]

    int32_t total_sum;                                                                      // totals
    int32_t total_sum2;
    int32_t total_count;

    // initialize sum, sum2, count, val buffers
    const int16_t ini = params.min_viable_number - 1;                                       // value that is NOT counted
    shared_sum[threadIdx.x] = 0;                                                            // shared values without effect on totals
    shared_sum2[threadIdx.x] = 0;
    shared_count[threadIdx.x] = 0;
    for (int i=0; i<window; i++)
        shared_val[i * blockDim.x + threadIdx.x] = ini;

    // wave front up to rmin + nby + 1
    do {
        if (data_col) {     // read at the front end of the wave
            const int16_t val = in[front * params.cols + col];
            shared_val[(front % window) * blockDim.x + threadIdx.x] = val;
            if (val >= params.min_viable_number) {
                shared_sum[threadIdx.x] += val;
                shared_sum2[threadIdx.x] += val * val;
                shared_count[threadIdx.x] += 1;
            }
        }
        front++;
    } while (front < rmin + static_cast<int32_t>(params.nby) + 1);
    // wave front up to rmax
    do {
        __syncthreads();    // make others see the shared values
        if (result_col) {   // write at the back end of the wave
            total_sum = total_sum2 = total_count = 0;
            for (auto j = left; j < right; j++) {
                total_sum += shared_sum[j];
                total_sum2 += shared_sum2[j];
                total_count += shared_count[j];
            }
            out[back * params.cols + col] = pixel_result(params, shared_val[(back % window) * blockDim.x + threadIdx.x], total_sum, total_sum2, total_count);
        }
        back++;
        __syncthreads();    // keep shared values until others have seen them
        if (data_col) {     // read at the front end of the wave
            int16_t cnt = 0;
            int16_t old = shared_val[(front % window) * blockDim.x + threadIdx.x];
            if (old < params.min_viable_number) {
                old = 0;    // no effect value
                cnt = 1;    // bring count to normal
            }
            int16_t val = in[front * params.cols + col];
            shared_val[(front % window) * blockDim.x + threadIdx.x] = val;
            if (val < params.min_viable_number) {
                val = 0;    // no effect value
                cnt -= 1;   // count diff from normal
            }
            shared_sum[threadIdx.x] += val - old;
            shared_sum2[threadIdx.x] += val * val - old * old;
            shared_count[threadIdx.x] += cnt;
        }
        front++;
    } while (front < rmax);
    // wave back up to rmax
    do {
        __syncthreads();    // make others see the shared values
        if (result_col) {   // write at the back end of the wave
            total_sum = total_sum2 = total_count = 0;
            for (auto j = left; j < right; j++) {
                total_sum += shared_sum[j];
                total_sum2 += shared_sum2[j];
                total_count += shared_count[j];
            }
            out[back * params.cols + col] = pixel_result(params, shared_val[(back % window) * blockDim.x + threadIdx.x], total_sum, total_sum2, total_count);
        }
        back++;
        __syncthreads();    // keep shared values until others have seen them
        if (data_col) {     // read at the front end of the wave if possible
            int16_t cnt = -1; // normal count diff
            int16_t old = shared_val[(front % window) * blockDim.x + threadIdx.x];
            if (old < params.min_viable_number) {
                old = 0;    // no effect value
                cnt += 1;   // bring count to normal
            }
            int16_t val = 0;
            if (front < params.lines) {
                val = in[front * params.cols + col];
                if (val < params.min_viable_number)
                    val = 0;    // no effect value
                else
                    cnt += 1;   // count diff from normal
            }
            shared_sum[threadIdx.x] += val - old;
            shared_sum2[threadIdx.x] += val * val - old * old;
            shared_count[threadIdx.x] += cnt;
        }
        front++;
    } while (back < rmax);
}

SpotFinder::SpotFinder(int32_t in_xpixels, int32_t in_ypixels, int32_t gpu_device) :
        xpixels(in_xpixels), ypixels(in_ypixels), gpu_out(nullptr) {

    int device_count;
    cuda_err(hipGetDeviceCount(&device_count));

    if (device_count == 0)
        throw JFJochException(JFJochExceptionCategory::GPUCUDAError, "No CUDA devices found");

    if (gpu_device < 0)
        gpu_device = threadid++;

    if (device_count > 1)
        cuda_err(hipSetDevice(gpu_device % device_count));

    int deviceId;
    cuda_err(hipGetDevice(&deviceId));
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    cudastream = new(CudaStreamWrapper);

    cuda_err(hipStreamCreate(&cudastream->v));
    cuda_err(hipMalloc(&gpu_in, xpixels * ypixels * sizeof(int16_t)));
    cuda_err(hipMalloc(&gpu_out, xpixels * ypixels * sizeof(char)));
    cuda_err(hipHostAlloc(&host_in_buffer, xpixels * ypixels * sizeof(int16_t),hipHostMallocPortable));
    cuda_err(hipHostAlloc(&host_out, xpixels * ypixels * sizeof(char), hipHostMallocPortable));
}

SpotFinder::SpotFinder(const DiffractionExperiment &experiment, int32_t gpu_device) :
        SpotFinder(experiment.GetXPixelsNum(), experiment.GetYPixelsNum(), gpu_device) {}

SpotFinder::~SpotFinder() {
    hipStreamDestroy(cudastream->v);
    delete(cudastream);

    hipHostFree(host_in_buffer);
    hipHostFree(host_out);
    hipFree(gpu_in);
    hipFree(gpu_out);
}

bool SpotFinder::GPUPresent() {
    int device_count;
    cuda_err(hipGetDeviceCount(&device_count));

    return (device_count > 0);
}

int16_t *SpotFinder::GetInputBuffer() {
    return host_in_buffer;
}

void SpotFinder::RunSpotFinder(const JFJochProtoBuf::DataProcessingSettings &settings) {
    // Run COLSPOT (GPU version)
    spot_parameters spot_params;
    spot_params.strong_pixel_threshold2 = settings.signal_to_noise_threshold() * settings.signal_to_noise_threshold();
    spot_params.nbx = settings.local_bkg_size();
    spot_params.nby = settings.local_bkg_size();
    spot_params.lines = ypixels;
    spot_params.cols = xpixels;
    spot_params.count_threshold = settings.photon_count_threshold();
    spot_params.min_viable_number = INT16_MIN + 5;

    if (2 * spot_params.nbx + 1 > windowSizeLimit)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "nbx exceeds window size limit");
    if (2 * spot_params.nby + 1 > windowSizeLimit)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "nby exceeds window size limit");
    if (windowSizeLimit > numberOfCudaThreads)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "window size limit exceeds number of cuda threads");
    if (windowSizeLimit > spot_params.cols)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "window size limit exceeds number of columns");
    if (windowSizeLimit > spot_params.lines)
        throw JFJochException(JFJochExceptionCategory::SpotFinderError, "window size limit exceeds number of lines");

    cuda_err(hipMemcpyAsync(gpu_in, host_in_buffer, xpixels * ypixels * sizeof(int16_t),
                             hipMemcpyHostToDevice, cudastream->v));

    { // call cuda kernel
        const auto nWriters = numberOfCudaThreads - 2 * spot_params.nby;
        const auto nBlocks = (spot_params.cols + nWriters - 1) / nWriters;
        const auto window = 2 * spot_params.nby + 1;
        const auto sharedSize = (2 * sizeof(int32_t) +            // sum, sum2
                                                      (1 + window) * sizeof(int16_t)   // count, val
                                                     ) * numberOfCudaThreads;
        const dim3 blocks(nBlocks, numberOfWaves);

        analyze_pixel<<<blocks, numberOfCudaThreads, sharedSize, cudastream->v>>>((int16_t *) gpu_in, gpu_out, spot_params);
    }

    cuda_err(hipMemcpyAsync(host_out, gpu_out, xpixels*ypixels*sizeof(char),
                             hipMemcpyDeviceToHost,cudastream->v));
}

void SpotFinder::GetResults(StrongPixelSet &pixel_set, int64_t image_number) {
    cuda_err(hipStreamSynchronize(cudastream->v));

    for (int line = 0; line < ypixels; line++) {
        for (int col = 0; col < xpixels; col++) {
            if (host_out[xpixels * line + col] > 0)
                pixel_set.AddStrongPixel(col, line, image_number, host_in_buffer[xpixels * line + col]);
        }
    }
}

void SpotFinder::GetResults(const DiffractionExperiment &experiment,
                            const JFJochProtoBuf::DataProcessingSettings &settings,
                            std::vector<DiffractionSpot> &vec,
                            int64_t image_number) {
    StrongPixelSet pixel_set;
    GetResults(pixel_set, image_number);
    pixel_set.FindSpots(experiment, settings, vec);
}

void SpotFinder::FindSpots(const DiffractionExperiment &experiment,
                           const JFJochProtoBuf::DataProcessingSettings &settings, std::vector<DiffractionSpot> &vec,
                           int64_t image_number) {
    RunSpotFinder(settings);
    GetResults(experiment, settings, vec, image_number);
}

std::atomic<uint16_t> SpotFinder::threadid{0};